#include "hip/hip_runtime.h"
#include "PCAPLoader.cuh"

// Load the pcap file into memory
std::vector<Packet> loadPcapIntoMemory(const std::string& filePath) {
    std::vector<Packet> packets;
    char errbuf[PCAP_ERRBUF_SIZE];

    // Open the pcap file
    pcap_t* handle = pcap_open_offline(filePath.c_str(), errbuf);
    if (handle == nullptr) {
        std::cerr << "Error opening file: " << errbuf << std::endl;
        exit(EXIT_FAILURE);
    }

    struct pcap_pkthdr* header;
    const u_char* data;
    int res, packetCount = 0;

    // Read packets from the file
    while ((res = pcap_next_ex(handle, &header, &data)) >= 0) {
        if (res == 0) {
            // Timeout, continue to next packet
            continue;
        }
        Packet packet;
        packet.data = data;
        packet.header = *header;
        packets.push_back(packet);
        packetCount++;
        if(packetCount == 100000) {
            break;
        }   
    }

    if (res == -1) {
        std::cerr << "Error reading packets: " << pcap_geterr(handle) << std::endl;
        pcap_close(handle);
        exit(EXIT_FAILURE);
    }

    std::cout << "Successfully loaded " << packets.size() << " packets into memory." << std::endl;

    // Close the pcap handle
    pcap_close(handle);
    return packets;
}

//Fragment the packet into multiple mbufs
struct rte_mbuf* createPacketFragmentChain(const u_char* data, uint32_t length, uint16_t fragment_size, struct rte_mempool* mbuf_pool) {
    struct rte_mbuf* head = nullptr;
    struct rte_mbuf* current = nullptr;
    uint16_t fragments = 0;

    while (length > 0) {
        uint16_t size = std::min(length, static_cast<uint32_t>(fragment_size));

        struct rte_mbuf* mbuf = rte_pktmbuf_alloc(mbuf_pool);
        if (!mbuf) {
            std::cerr << "Failed to allocate mbuf for packet fragment." << std::endl;
            // Free any previously allocated fragments
            while (head) {
                struct rte_mbuf* temp = head->next;
                rte_pktmbuf_free(head);
                head = temp;
            }
            return nullptr;
        }

        uint8_t* pkt_data = (uint8_t*)rte_pktmbuf_append(mbuf, size);
        if (!pkt_data) {
            std::cout << "Failed to append data to mbuf." << std::endl;
            rte_pktmbuf_free(mbuf);
            while (head) {
                struct rte_mbuf* temp = head->next;
                rte_pktmbuf_free(head);
                head = temp;
            }
            return nullptr;
        }

        memcpy(pkt_data, data, size);
        data += size;
        length -= size;

        if (!head) {
            head = mbuf; // First fragment
        } else {
            current->next = mbuf; // Link previous fragment
        }
        current = mbuf;
        fragments++;
    }
    
    return head;
}